#include "hip/hip_runtime.h"
#define USE_MNIST_LOADER
#define MNIST_DOUBLE
#include "mnist.h"
#include "layer.h"

#include <hip/hip_runtime.h>
#include <cstdio>
#include <time.h>

static mnist_data *train_set, *test_set;
static unsigned int train_cnt, test_cnt;

// Define layers of CNN
static Layer l_input = Layer(0, 0, 28*28);
static Layer l_c1 = Layer(5*5, 6, 24*24*6);
static Layer l_c2 = Layer(2*2, 6, 12*12*6);
static Layer l_c3 = Layer(2*2, 6, 6*6*6);
static Layer l_f = Layer(6*6*6, 10, 10);

//resnet shortcut
static Layer l_r = Layer(4*4,1,6*6*6);

static void learn();
static unsigned int classify(double data[28][28]);
static void test();
static double forward_pass(double data[28][28]);
static double back_pass();

static inline void loaddata()
{
	mnist_load("data/train-images.idx3-ubyte", "data/train-labels.idx1-ubyte",
		&train_set, &train_cnt);
	mnist_load("data/t10k-images.idx3-ubyte", "data/t10k-labels.idx1-ubyte",
		&test_set, &test_cnt);
}

int main(int argc, const  char **argv)
{
	srand(time(NULL));

	hipError_t err = hipInit(0);
	if (err != hipSuccess) {
		fprintf(stderr, "CUDA initialisation failed with error code - %d\n", err);
		return 1;
	}

	loaddata();
	learn();
	test();

	return 0;
}

// Forward propagation of a single row in dataset
static double forward_pass(double data[28][28])
{
	float input[28][28];

	for (int i = 0; i < 28; ++i) {
		for (int j = 0; j < 28; ++j) {
			input[i][j] = data[i][j];
		}
	}

	l_input.clear();
	l_c1.clear();
	l_c2.clear();
	l_c3.clear();
	l_f.clear();
	l_r.clear();
	clock_t start, end;
	start = clock();

	l_input.setOutput((float *)input);
	
	fp_preact_c1<<<128, 128>>>((float (*)[28])l_input.output, (float (*)[24][24])l_c1.preact, (float (*)[5][5])l_c1.weight);
	fp_bias_c1<<<128, 128>>>((float (*)[24][24])l_c1.preact, l_c1.bias);
	apply_sigmoid<<<128, 128>>>(l_c1.preact, l_c1.output, l_c1.O);

	fp_preact_r<<<128, 128>>>((float (*)[24][24])l_c1.preact, (float (*)[6][6])l_r.preact, (float (*)[4][4])l_r.weight);
	fp_bias_r<<<128, 128>>>((float (*)[6][6])l_r.preact, l_r.bias);

	fp_preact_c2<<<128, 128>>>((float (*)[24][24])l_c1.output, (float (*)[12][12])l_c2.preact, (float (*)[2][2])l_c2.weight);
	fp_bias_c2<<<128, 128>>>((float (*)[12][12])l_c2.preact, l_c2.bias);
	apply_sigmoid<<<128, 128>>>(l_c2.preact, l_c2.output, l_c2.O);

	fp_preact_c3<<<128, 128>>>((float (*)[12][12])l_c2.output, (float (*)[6][6])l_c3.preact, (float (*)[2][2])l_c3.weight);
	fp_bias_c3<<<128, 128>>>((float (*)[6][6])l_c3.preact, l_c3.bias);

	fp_add_res<<<128, 128>>>((float (*)[6][6])l_c3.preact, (float (*)[6][6])l_r.preact);
	
	apply_sigmoid<<<128, 128>>>(l_c3.preact, l_c3.output, l_c3.O);
	

	fp_preact_f<<<128, 128>>>((float (*)[6][6])l_c3.output, l_f.preact, (float (*)[6][6][6])l_f.weight);
	fp_bias_f<<<128, 128>>>(l_f.preact, l_f.bias);
	apply_sigmoid<<<128, 128>>>(l_f.preact, l_f.output, l_f.O);
	
	end = clock();
	return ((double) (end - start)) / CLOCKS_PER_SEC;
}

// Back propagation to update weights
static double back_pass()
{
	clock_t start, end;

	start = clock();

	bp_weight_f<<<128, 128>>>((float (*)[6][6][6])l_f.d_weight, l_f.d_preact, (float (*)[6][6])l_c3.output);
	bp_bias_f<<<128, 128>>>(l_f.bias, l_f.d_preact);

	bp_output_c3<<<128, 128>>>((float (*)[6][6])l_c3.d_output, (float (*)[6][6][6])l_f.weight, l_f.d_preact);
	bp_preact_c3<<<128, 128>>>((float (*)[6][6])l_c3.d_preact, (float (*)[6][6])l_c3.d_output, (float (*)[6][6])l_c3.preact);
	bp_weight_c3<<<128, 128>>>((float (*)[2][2])l_c3.d_weight, (float (*)[6][6])l_c3.d_preact, (float (*)[12][12])l_c2.output);
	bp_bias_c3<<<128, 128>>>(l_c3.bias, (float (*)[6][6])l_c3.d_preact);

	bp_output_c2<<<128, 128>>>((float (*)[12][12])l_c2.d_output, (float (*)[2][2])l_c3.weight, (float (*)[6][6])l_c3.d_preact);
	bp_preact_c2<<<128, 128>>>((float (*)[12][12])l_c2.d_preact, (float (*)[12][12])l_c2.d_output, (float (*)[12][12])l_c2.preact);
	bp_weight_c2<<<128, 128>>>((float (*)[2][2])l_c2.d_weight, (float (*)[12][12])l_c2.d_preact, (float (*)[24][24])l_c1.output);
	bp_bias_c2<<<128, 128>>>(l_c2.bias, (float (*)[12][12])l_c2.d_preact);

	bp_output_c1<<<128, 128>>>((float (*)[24][24])l_c1.d_output, (float (*)[2][2])l_c2.weight, (float (*)[12][12])l_c2.d_preact);
	bp_preact_c1<<<128, 128>>>((float (*)[24][24])l_c1.d_preact, (float (*)[24][24])l_c1.d_output, (float (*)[24][24])l_c1.preact);
	bp_weight_c1<<<128, 128>>>((float (*)[5][5])l_c1.d_weight, (float (*)[24][24])l_c1.d_preact, (float (*)[28])l_input.output);
	bp_bias_c1<<<128, 128>>>(l_c1.bias, (float (*)[24][24])l_c1.d_preact);


	apply_grad<<<128, 128>>>(l_f.weight, l_f.d_weight, l_f.M * l_f.N);
	apply_grad<<<128, 128>>>(l_c2.weight, l_c2.d_weight, l_c2.M * l_c2.N);
	apply_grad<<<128, 128>>>(l_c1.weight, l_c1.d_weight, l_c1.M * l_c1.N);

	end = clock();
	return ((double) (end - start)) / CLOCKS_PER_SEC;
}

// Unfold the input layer
static void unfold_input(double input[28][28], double unfolded[24*24][5*5])
{
	int a = 0;
	(void)unfold_input;

	for (int i = 0; i < 2; ++i)
		for (int j = 0; j < 2; ++j) {
			int b = 0;
			for (int x = i; x < i + 2; ++x)
				for (int y = j; y < j+2; ++y)
					unfolded[a][b++] = input[x][y];
			a++;
		}
}

static void learn()
{
	static hipblasHandle_t blas;
	hipblasCreate(&blas);

	float err;
	int iter = 20;
	
	double time_taken = 0.0;

	fprintf(stdout ,"Learning\n");

	while (iter < 0 || iter-- > 0) {
		err = 0.0f;

		for (int i = 0; i < train_cnt; ++i) {
			float tmp_err;

			time_taken += forward_pass(train_set[i].data);

			l_f.bp_clear();
			l_c2.bp_clear();
			l_c1.bp_clear();
			l_c3.bp_clear();
			// Euclid distance of train_set[i]
			makeError<<<10, 1>>>(l_f.d_preact, l_f.output, train_set[i].label, 10);
			hipblasSnrm2(blas, 10, l_f.d_preact, 1, &tmp_err);
			err += tmp_err;

			time_taken += back_pass();
		}

		err /= train_cnt;
		double accuracy = 100 - double(err) * 100.0;
		fprintf(stdout, "accuracy: %.2lf%% , time_on_gpu: %lf sec\n", accuracy, time_taken);

		if (err < threshold) {
			fprintf(stdout, "Training complete, error less than threshold\n\n");
			break;
		}

	}
	
	fprintf(stdout, "\n Time - %lf s\n", time_taken);
}


// Returns label of given data (0-9)
static unsigned int classify(double data[28][28])
{
	float res[10];

	forward_pass(data);

	unsigned int max = 0;

	hipMemcpy(res, l_f.output, sizeof(float) * 10, hipMemcpyDeviceToHost);

	for (int i = 1; i < 10; ++i) {
		if (res[max] < res[i]) {
			max = i;
		}
	}

	return max;
}

// Perform forward propagation of test data
static void test()
{
	int error = 0;

	for (int i = 0; i < test_cnt; ++i) {
		if (classify(test_set[i].data) != test_set[i].label) {
			++error;
		}
	}
	double err_percent = double(error) / double(test_cnt) * 100.0;
	fprintf(stdout, "Error Rate: %.2lf%% , accuracy: %.2lf%%\n",err_percent,100-err_percent);
}
